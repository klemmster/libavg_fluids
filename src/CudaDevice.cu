/*
 * CudaCard.cpp
 *
 *  Created on: May 30, 2012
 *      Author: andyf
 */

#include "CudaDevice.h"

namespace avg
{


CudaDevice::CudaDevice() {

	hipError_t error;

	error = hipInit(0);
	if (error != hipSuccess){
		printf("cuda init error");
	}
	int deviceCount = 0;
	error = hipGetDeviceCount(&deviceCount);
	if (error != hipSuccess){
		throw std::runtime_error("Failed to get CudaDeviceCount.\n");
	}

	if (deviceCount == 0) {
		throw std::runtime_error("There is no device supporting CUDA.\n");
	}
	int devID=0;
	m_cuDevice = new hipDevice_t();
	error = hipDeviceGet(m_cuDevice, devID);
	if (error != hipSuccess){
		throw std::runtime_error("could not get device\n");
	}
    error = cuGLCtxCreate(m_cuContext, hipDeviceScheduleAuto, *m_cuDevice);
	if (error != hipSuccess){
        std::cout << "Error: " << error << "\n";
		throw std::runtime_error("could not create cudaGLContext\n");
	}
    checkCudaErrors(cudaGLSetGLDevice(*m_cuDevice));
}

CudaDevice::~CudaDevice() {
    std::cout << "Destroy Cuda Device\n";
    hipError_t error;
    error = hipCtxDestroy(*m_cuContext);
    if (error != hipSuccess){
        std::cout << "Context not properly destroyed\n";
    }
    delete m_cuDevice;
    hipDeviceReset();
}


hipDevice_t * CudaDevice::getDevice() {
	return m_cuDevice;
}

} /* avg */

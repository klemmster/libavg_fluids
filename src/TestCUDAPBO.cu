#include "hip/hip_runtime.h"
#include "TestCUDAPBO.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cutil_inline.h>
#include <iostream>

extern "C" __global__
void testPBO( char* dst){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x > 0 && x < 512)
        if(y > 0  && y < 512){
        int index = y*512*4 + x;
            dst[index] = (char)128;
        }
}

namespace avg
{

TestCUDAPBO::TestCUDAPBO()
{
}

TestCUDAPBO::~TestCUDAPBO(){

}

void TestCUDAPBO::step(){

    checkCudaErrors(hipGraphicsMapResources( 1, &m_cuPBO));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer( (void**)&m_devPtr, &m_size, m_cuPBO));

    /*
    std::cout << m_devPtr << "\n";
    std::cout << m_size << "\n";
    */

    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(1, 1, 1);
    testPBO<<< dimGrid, dimBlock >>>((char*)m_devPtr);
    checkCudaErrors(hipGetLastError());
    hipGraphicsUnmapResources(1, &m_cuPBO);
}

void TestCUDAPBO::setPBO(unsigned pbo){
    m_pbo = pbo;
    checkCudaErrors(hipGraphicsGLRegisterBuffer( &m_cuPBO, pbo, cudaGraphicsMapFlagsNone));
}

} /* avg */


